#include "hip/hip_runtime.h"
/* Blur filter. Device code. */

#ifndef _BLUR_FILTER_KERNEL_H_
#define _BLUR_FILTER_KERNEL_H_

#include "blur_filter.h"

__global__ void 
blur_filter_kernel (const float *in, float *out, int size)
{
	int threadX = threadIdx.x;
	int threadY = threadIdx.y;
	int blockX = blockIdx.x;
	int blockY = blockIdx.y;
	int row = blockDim.y * blockY + threadY;
	int col = blockDim.x * blockX + threadX;
	int i, j, num_neighbors, pixelIndex, curr_row, curr_col;
	float blur_value;
	pixelIndex = row * size + col;
	blur_value = 0.0;
	num_neighbors = 0;
	for(i = -BLUR_SIZE; i < (BLUR_SIZE + 1); i++)
	{
		for(j = -BLUR_SIZE; j < (BLUR_SIZE + 1); j++)
		{
			curr_row = row + i;
			curr_col = col + j;
			if((curr_row > -1) && (curr_row < size) &&\
				(curr_col > -1) && (curr_col < size))
			{
				blur_value += in[curr_row * size + curr_col];
				num_neighbors += 1;
			}
		}
	}
	out[pixelIndex] = blur_value/num_neighbors;
		
    	return;
}

#endif /* _BLUR_FILTER_KERNEL_H_ */
