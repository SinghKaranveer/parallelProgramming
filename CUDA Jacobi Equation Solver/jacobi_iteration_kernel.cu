#include "hip/hip_runtime.h"
#include "jacobi_iteration.h"

/* FIXME: Write the device kernels to solve the Jacobi iterations. */


__global__ void jacobi_iteration_kernel_naive (const float *A, float *new_naive_cuda_x, float *x, const float *B, unsigned int num_rows, unsigned int num_cols)
{
	int threadX = threadIdx.x;
	int threadY = threadIdx.y;
	int blockX = blockIdx.x;
	int blockY = blockIdx.y;
	int row = blockDim.y * blockY + threadY;
	int col = blockDim.x * blockX + threadX;
	double sum;
	int i, j;

	printf("\nFor Loop I starts\n");
        for (i = 0; i < num_rows; i++){
             double sum = -A[i * num_cols + i] * x[i];
	printf("\nFor Loop J starts\n");
             for (j = 0; j < num_cols; j++)
                 sum += A[i * num_cols + j] * x[j];
        }
       printf("\nData written\n");
        new_naive_cuda_x[i] = (B[i] - sum)/A[i * num_cols + i];
           
	return;
}

__global__ void jacobi_iteration_kernel_optimized ()
{
    return;
}

